#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 512

extern "C" {
__global__ void lagrangeUpdate(int ie, int je, int k, int klocal, \
         float orelax, float va2s, int dim_1,  int dim_2, int dim_3, int tnz, \
         float* ve,float* vf,float* vg,float* vh,float* vm,float* vn, \
         float* va2f,float* vr,float* vd,float* vp1,float* t_p1)
        {
	 int i = blockDim.x * blockIdx.x + threadIdx.x;
	 int j = blockDim.y * blockIdx.y + threadIdx.y;

	 int ijk=k*(dim_1*dim_2)+j*dim_1+i  ;
         int ip1jk=k*(dim_1*dim_2)+j*dim_1+(i+1)    ;
         int im1jk=k*(dim_1*dim_2)+j*dim_1+(i-1)    ;
         int ijp1k=k*(dim_1*dim_2)+(j+1)*dim_1+i    ;
         int ijm1k=k*(dim_1*dim_2)+(j-1)*dim_1+i    ;
         int ijkp1=(k+1)*(dim_1*dim_2)+j*dim_1+i    ;
         int ijkm1=(k-1)*(dim_1*dim_2)+j*dim_1+i    ;
         int ijkl=klocal*(dim_1*dim_2)+j*dim_1+i    ;

	 if (i < ie && j < je){
                t_p1[ijkl]=vd[ijk]*(ve[ijk]*vp1[ip1jk]+\
                      vf[ijk]*vp1[im1jk]+            \
                      vg[ijk]*vp1[ijp1k]+            \
                      vh[ijk]*vp1[ijm1k]+            \
                      (vm[ijk]*vp1[ijkp1]+           \
                      vn[ijk]*vp1[ijkm1])*           \
                      va2s / (va2f[ijk]*va2f[ijk])-  \
                      vr[ijk])+ orelax*vp1[ijk] ;
		}
	}


void sor3d_flat_fire_comp( int is, int ie, int js, \
	 int je, int k, int klocal, int step, \
         float orelax,  int dim_1,  int dim_2, int dim_3, int tnz, \
         float* ve,float* vf,float* vg,float* vh,float* vm,float* vn, \
	 float va2s,float* va2f,float* vr,float* vd,float* vp1,float* t_p1)
	{

	js=js-1;	
	is=is-1;	
	k=k-1;	
	klocal=klocal-1;	

	int size=dim_1*dim_2*dim_3*sizeof(float);
	int sizem1=dim_1*dim_2*(dim_3-1)*sizeof(float);
	int sizet1=dim_1*dim_2*tnz*sizeof(float);

	float *dd; 
	float *de; 
	float *df; 
	float *dg; 
	float *dh; 
	float *dm; 
	float *dn; 
	float *da2f; 
	float *dr; 
	float *dp1; 
	float *dtp1; 

	hipMalloc((void **)&dd, sizem1);
	hipMalloc((void **)&de, sizem1);
	hipMalloc((void **)&df, sizem1);
	hipMalloc((void **)&dg, sizem1);
	hipMalloc((void **)&dh, sizem1);
	hipMalloc((void **)&dm, sizem1);
	hipMalloc((void **)&dn, sizem1);
	hipMalloc((void **)&da2f, size);
	hipMalloc((void **)&dr, sizem1);
	hipMalloc((void **)&dp1, size);
	hipMalloc((void **)&dtp1, sizet1);



	hipMemcpy(dd, vd, sizem1, hipMemcpyHostToDevice);
	hipMemcpy(de, ve, sizem1, hipMemcpyHostToDevice);
	hipMemcpy(df, vf, sizem1, hipMemcpyHostToDevice);
	hipMemcpy(dg, vg, sizem1, hipMemcpyHostToDevice);
	hipMemcpy(dh, vh, sizem1, hipMemcpyHostToDevice);
	hipMemcpy(dm, vm, sizem1, hipMemcpyHostToDevice);
	hipMemcpy(dn, vn, sizem1, hipMemcpyHostToDevice);
	hipMemcpy(dr, vr, sizem1, hipMemcpyHostToDevice);
	hipMemcpy(da2f, va2f, size, hipMemcpyHostToDevice);
	hipMemcpy(dp1, vp1, size, hipMemcpyHostToDevice);

	dim3 dblock(BLOCK_DIM,BLOCK_DIM);
  	dim3 dgrid(dim_1 / dblock.x, dim_2 / dblock.y);

	lagrangeUpdate<<<dgrid, dblock>>>(ie,je,k,klocal,orelax,va2s,dim_1,dim_2,dim_3,tnz,de,df,dg,dh,dm,dn,da2f,dr,dd,dp1,dtp1);

	hipMemcpy(t_p1, dtp1, size, hipMemcpyDeviceToHost);
}

}
